#include "hip/hip_runtime.h"
#include <config.h>

#define BOOST_TEST_MODULE TestGpuCo2GasPvt

#include <boost/test/unit_test.hpp>
#include <opm/material/common/MathToolbox.hpp>
#include <opm/material/densead/Evaluation.hpp>
#include <opm/material/densead/Math.hpp>
#include <opm/material/common/UniformTabulated2DFunction.hpp>
#include <opm/material/components/CO2Tables.hpp>
#include <opm/material/components/CO2.hpp>
#include <opm/material/components/SimpleHuDuanH2O.hpp>
#include <opm/material/components/BrineDynamic.hpp>
#include <opm/material/binarycoefficients/Brine_CO2.hpp>
#include <opm/material/fluidsystems/blackoilpvt/Co2GasPvt.hpp>
#include <opm/material/fluidsystems/blackoilpvt/BrineCo2Pvt.hpp>
#include <opm/input/eclipse/EclipseState/Co2StoreConfig.hpp>

#include <opm/simulators/linalg/gpuistl/detail/gpu_safe_call.hpp>
#include <opm/simulators/linalg/gpuistl/GpuBuffer.hpp>
#include <opm/simulators/linalg/gpuistl/GpuView.hpp>

#include <hip/hip_runtime.h>
#include <vector>
#include <utility>
#include <cmath> // For tolerance-based floating-point comparison

using Evaluation = Opm::DenseAd::Evaluation<float, 3>;
using GpuB = const Opm::gpuistl::GpuBuffer<double>;
using GpuV = Opm::gpuistl::GpuView<const double>;

using GpuTab = Opm::UniformTabulated2DFunction<double, GpuV>;

using GpuBufCo2Tables = Opm::CO2Tables<double, GpuB>;
using GpuViewCO2Tables = Opm::CO2Tables<double, GpuV>;
template Opm::CO2Tables<double, GpuB>::CO2Tables(const Opm::UniformTabulated2DFunction<double, GpuB>&, const Opm::UniformTabulated2DFunction<double, GpuB>&);
template Opm::CO2Tables<double, GpuV>::CO2Tables(const Opm::UniformTabulated2DFunction<double, GpuV>&, const Opm::UniformTabulated2DFunction<double, GpuV>&);
using GpuCO2 = Opm::CO2<double, GpuViewCO2Tables>;

using HuDuan = Opm::SimpleHuDuanH2O<double>;
using BrineDyn = Opm::BrineDynamic<double, HuDuan>;

using CpuBrine_CO2 = Opm::BinaryCoeff::Brine_CO2<double, HuDuan, Opm::CO2<double>>;
using GpuBrine_CO2 = Opm::BinaryCoeff::Brine_CO2<double, HuDuan, GpuCO2>;

using CpuCo2Pvt = Opm::Co2GasPvt<double>;
using GpuBufCo2Pvt = Opm::Co2GasPvt<double, GpuBufCo2Tables, GpuB>;
using GpuViewCo2Pvt = Opm::Co2GasPvt<double, GpuViewCO2Tables, GpuV>;

template GpuBufCo2Pvt::Co2GasPvt(GpuBufCo2Tables, GpuB, GpuB, GpuB, bool, bool, int, Co2StoreConfig::GasMixingType);
template GpuViewCo2Pvt::Co2GasPvt(GpuViewCO2Tables, GpuV, GpuV, GpuV, bool, bool, int, Co2StoreConfig::GasMixingType);

using CpuBrineCo2Pvt = Opm::BrineCo2Pvt<double>;
using GpuBufBrineCo2Pvt = Opm::BrineCo2Pvt<double, GpuBufCo2Tables, GpuB>;
using GpuViewBrineCo2Pvt = Opm::BrineCo2Pvt<double, GpuViewCO2Tables, GpuV>;

template GpuBufBrineCo2Pvt::BrineCo2Pvt(GpuB, GpuB, GpuB, int, int, int, GpuBufCo2Tables);
template GpuViewBrineCo2Pvt::BrineCo2Pvt(GpuV, GpuV, GpuV, int, int, int, GpuViewCO2Tables);
namespace {

/*
    This file containts tests using Co2GasPvt on the GPU in addition to helper objects
*/
// TODO: Rewrite these tests using a fixture pattern to greatly reduce the code duplication

// struct Fixture {
//     Fixture(){
//         double viscosity = Opm::CO2<double>::gasViscosity<Evaluation>(temp, pressure, true).value();

//     // make a nonstatic version of the CPU CO2
//     Opm::CO2NonStatic<double> CO2(Opm::CO2<double>::getEnthalpy(), Opm::CO2<double>::getDensity());

//     const auto gpuEnthalpyBuffer = Opm::gpuistl::move_to_gpu<double, GpuB>(CO2.getEnthalpy());
//     const auto gpuDensityBuffer = Opm::gpuistl::move_to_gpu<double, GpuB>(CO2.getDensity());

//     const auto gpuEnthalpyView = Opm::gpuistl::make_view<double, GpuB, GpuV>(gpuEnthalpyBuffer);
//     const auto gpuDensityView = Opm::gpuistl::make_view<double, GpuB, GpuV>(gpuDensityBuffer);
//     }
//     ~Fixture(){

//     }
// };

// Kernel to evaluate a 2D function on the GPU
__global__ void gpuEvaluateUniformTabulated2DFunction(GpuTab gpuTab, Evaluation* inputX, Evaluation* inputY, double* res) {
    *res = gpuTab.eval(*inputX, *inputY, true).value();
}

} // END EMPTY NAMESPACE

// Test case for evaluating a tabulated 2D function on both CPU and GPU
BOOST_AUTO_TEST_CASE(TestEvaluateUniformTabulated2DFunctionOnGpu) {
    // Example tabulated data (2D)
    std::vector<std::vector<double>> tabData = {{1.0, 2.0}, {3.0, 4.0}, {5.0, 6.0}};

    // CPU-side function definition
    Opm::UniformTabulated2DFunction<double> cpuTab(1.0, 6.0, 3, 1.0, 6.0, 2, tabData);

    // Move data to GPU buffer and create a view for GPU operations
    Opm::UniformTabulated2DFunction<double, GpuB> gpuBufTab = Opm::gpuistl::move_to_gpu<double, GpuB>(cpuTab);
    GpuTab gpuViewTab = Opm::gpuistl::make_view<double, GpuB, GpuV>(gpuBufTab);

    // Evaluation points on the CPU
    Evaluation a(2.3);
    Evaluation b(4.5);

    // Allocate memory for the result on the GPU
    double* resultOnGpu = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&resultOnGpu, sizeof(double)));

    // Allocate GPU memory for the Evaluation inputs
    Evaluation* gpuA = nullptr;
    Evaluation* gpuB = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuA, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuA, &a, sizeof(Evaluation), hipMemcpyHostToDevice));
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuB, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuB, &b, sizeof(Evaluation), hipMemcpyHostToDevice));

    // Launch kernel to evaluate the function on the GPU
    gpuEvaluateUniformTabulated2DFunction<<<1, 1>>>(gpuViewTab, gpuA, gpuB, resultOnGpu);

    // Check for any errors in kernel launch
    OPM_GPU_SAFE_CALL(hipPeekAtLastError());
    OPM_GPU_SAFE_CALL(hipDeviceSynchronize());

    // Retrieve the result from the GPU to the CPU
    double resultOnCpu = 0.0;
    OPM_GPU_SAFE_CALL(hipMemcpy(&resultOnCpu, resultOnGpu, sizeof(double), hipMemcpyDeviceToHost));

    // Free allocated GPU memory
    OPM_GPU_SAFE_CALL(hipFree(resultOnGpu));
    OPM_GPU_SAFE_CALL(hipFree(gpuA));
    OPM_GPU_SAFE_CALL(hipFree(gpuB));

    // Verify that the CPU and GPU results match within a reasonable tolerance
    const double cpuResult = cpuTab.eval(a, b, true).value();
    const double tolerance = 1e-6; // Tolerance for floating-point comparison
    BOOST_CHECK(std::fabs(resultOnCpu - cpuResult) < tolerance);
}


namespace {

// Kernel to use a CO2 object on the GPU
__global__ void gpuCO2(GpuViewCO2Tables gpuViewCo2Tables, Evaluation* temp, Evaluation* pressure, double* resultViscosity) {
    *resultViscosity = GpuCO2::gasViscosity<Evaluation>(gpuViewCo2Tables, *temp, *pressure, true).value();
}

} // END EMPTY NAMESPACE

// Test case evaluating CO2 pvt properties on CPU and GPU
BOOST_AUTO_TEST_CASE(TestUseCO2OnGpu) {
    Evaluation temp(290.5); // [K]
    Evaluation pressure(200000.0); // [Pa]

    // make a nonstatic version of the CPU CO2tables
    Opm::CO2Tables<double, std::vector<double>> co2Tables;

    // use the CO2 tables to aquire the viscosity at 290[K] and 2e5[Pa]
    double viscosity = Opm::CO2<double, Opm::CO2Tables<double, std::vector<double>>>::gasViscosity<Evaluation>(co2Tables, temp, pressure, true).value();

    GpuBufCo2Tables gpuBufCo2Table = Opm::gpuistl::move_to_gpu<double, std::vector<double>, GpuB>(co2Tables);
    GpuViewCO2Tables gpuViewCo2Table = Opm::gpuistl::make_view<double, GpuB, GpuV>(gpuBufCo2Table);
    // GpuCO2 gpuCo2(gpuEnthalpyView, gpuDensityView);

    // Allocate memory for the result on the GPU
    double* resultOnGpu = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&resultOnGpu, sizeof(double)));

    // Allocate GPU memory for the Evaluation inputs
    Evaluation* gpuTemp = nullptr;
    Evaluation* gpuPressure = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuTemp, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuTemp, &temp, sizeof(Evaluation), hipMemcpyHostToDevice));
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuPressure, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuPressure, &pressure, sizeof(Evaluation), hipMemcpyHostToDevice));

    gpuCO2<<<1,1>>>(gpuViewCo2Table, gpuTemp, gpuPressure, resultOnGpu);

    // Check for any errors in kernel launch
    OPM_GPU_SAFE_CALL(hipPeekAtLastError());
    OPM_GPU_SAFE_CALL(hipDeviceSynchronize());

    // Retrieve the result from the GPU to the CPU
    double resultOnCpu = 0.0;
    OPM_GPU_SAFE_CALL(hipMemcpy(&resultOnCpu, resultOnGpu, sizeof(double), hipMemcpyDeviceToHost));

    // Free allocated GPU memory
    OPM_GPU_SAFE_CALL(hipFree(resultOnGpu));
    OPM_GPU_SAFE_CALL(hipFree(gpuTemp));
    OPM_GPU_SAFE_CALL(hipFree(gpuPressure));

    // Verify that the CPU and GPU results match within a reasonable tolerance
    const double tolerance = 1e-6; // Tolerance for floating-point comparison
    BOOST_CHECK(std::fabs(resultOnCpu - viscosity) < tolerance);
}


namespace {

// Kernel to use a SimpleHuDuanH20 object on a GPU
__global__ void liquidDensity(Evaluation* temp, Evaluation* pressure, double* resultDensity) {
    *resultDensity = HuDuan::liquidDensity<Evaluation>(*temp, *pressure, true).value();
}

} // END EMPTY NAMESPACE

// Test case evaluating pvt values for SimpleHuDuanH20 on a GPU and CPU
BOOST_AUTO_TEST_CASE(TestUseH2OOnGpu) {
    Evaluation temp(290.5); // [K]
    Evaluation pressure(200000.0); // [Pa]

    // use the CO2 tables to aquire the densityReference at 290[K] and 2e5[Pa]
    double densityReference = HuDuan::liquidDensity<Evaluation>(temp, pressure, true).value();

    // Allocate memory for the result on the GPU
    double* resultOnGpu = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&resultOnGpu, sizeof(double)));

    // Allocate GPU memory for the Evaluation inputs
    Evaluation* gpuTemp = nullptr;
    Evaluation* gpuPressure = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuTemp, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuTemp, &temp, sizeof(Evaluation), hipMemcpyHostToDevice));
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuPressure, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuPressure, &pressure, sizeof(Evaluation), hipMemcpyHostToDevice));

    liquidDensity<<<1,1>>>(gpuTemp, gpuPressure, resultOnGpu);

    // Check for any errors in kernel launch
    OPM_GPU_SAFE_CALL(hipPeekAtLastError());
    OPM_GPU_SAFE_CALL(hipDeviceSynchronize());

    // Retrieve the result from the GPU to the CPU
    double resultOnCpu = 0.0;
    OPM_GPU_SAFE_CALL(hipMemcpy(&resultOnCpu, resultOnGpu, sizeof(double), hipMemcpyDeviceToHost));

    // Free allocated GPU memory
    OPM_GPU_SAFE_CALL(hipFree(resultOnGpu));
    OPM_GPU_SAFE_CALL(hipFree(gpuTemp));
    OPM_GPU_SAFE_CALL(hipFree(gpuPressure));

    // Verify that the CPU and GPU results match within a reasonable tolerance
    const double tolerance = 1e-6; // Tolerance for floating-point comparison
    BOOST_CHECK(std::fabs(resultOnCpu - densityReference) < tolerance);
}


namespace {

// Kernel to use a BrineDynamic object on a GPU
__global__ void liquidEnthalpy(Evaluation* temp, Evaluation* pressure, Evaluation* salinity, double* resultEnthalpy) {
    *resultEnthalpy = BrineDyn::liquidEnthalpy<Evaluation>(*temp, *pressure, *salinity).value();
}

} // END EMPTY NAMESPACE

// Test case evaluating pvt values for BrineDynamic on a GPU and CPU
BOOST_AUTO_TEST_CASE(TestUseBrineDynamicOnGpu) {
    Evaluation temp(290.5); // [K]
    Evaluation pressure(200000.0); // [Pa]
    Evaluation salinity(0.1); // [g/Kg]

    // use the CO2 tables to aquire the enthalpyReference at 290[K] and 2e5[Pa]
    double enthalpyReference = BrineDyn::liquidEnthalpy<Evaluation>(temp, pressure, salinity).value();

    // Allocate memory for the result on the GPU
    double* resultOnGpu = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&resultOnGpu, sizeof(double)));

    // Allocate GPU memory for the Evaluation inputs
    Evaluation* gpuTemp = nullptr;
    Evaluation* gpuPressure = nullptr;
    Evaluation* gpuSalinity = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuTemp, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuTemp, &temp, sizeof(Evaluation), hipMemcpyHostToDevice));
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuPressure, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuPressure, &pressure, sizeof(Evaluation), hipMemcpyHostToDevice));
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuSalinity, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuSalinity, &salinity, sizeof(Evaluation), hipMemcpyHostToDevice));

    liquidEnthalpy<<<1,1>>>(gpuTemp, gpuPressure, gpuSalinity, resultOnGpu);

    // Check for any errors in kernel launch
    OPM_GPU_SAFE_CALL(hipPeekAtLastError());
    OPM_GPU_SAFE_CALL(hipDeviceSynchronize());

    // Retrieve the result from the GPU to the CPU
    double resultOnCpu = 0.0;
    OPM_GPU_SAFE_CALL(hipMemcpy(&resultOnCpu, resultOnGpu, sizeof(double), hipMemcpyDeviceToHost));

    // Free allocated GPU memory
    OPM_GPU_SAFE_CALL(hipFree(resultOnGpu));
    OPM_GPU_SAFE_CALL(hipFree(gpuTemp));
    OPM_GPU_SAFE_CALL(hipFree(gpuPressure));

    // Verify that the CPU and GPU results match within a reasonable tolerance
    const double tolerance = 1e-6; // Tolerance for floating-point comparison
    BOOST_CHECK(std::fabs(resultOnCpu - enthalpyReference) < tolerance);
}


namespace {

// Kernel to use a BrineDynamic object on a GPU
__global__ void gasDiffCoeff(GpuViewCO2Tables co2tables, Evaluation* temp, Evaluation* pressure, double* result) {
    *result = GpuBrine_CO2::gasDiffCoeff<Evaluation, GpuViewCO2Tables>(co2tables, *temp, *pressure, true).value();
}

} // END EMPTY NAMESPACE

// Test case evaluating pvt values for BrineDynamic on a GPU and CPU
BOOST_AUTO_TEST_CASE(TestBrine_CO2OnGPU) {
    Evaluation temp(290.5); // [K]
    Evaluation pressure(200000.0); // [Pa]


    // make a nonstatic version of the CPU CO2tables
    Opm::CO2Tables<double, std::vector<double>> co2Tables;

    // use the CO2 tables to aquire the enthalpyReference at 290[K] and 2e5[Pa]
    double enthalpyReference = CpuBrine_CO2::gasDiffCoeff<Evaluation>(co2Tables, temp, pressure, true).value();

    // use the CO2 tables to aquire the viscosity at 290[K] and 2e5[Pa]
    double viscosity = Opm::CO2<double, Opm::CO2Tables<double, std::vector<double>>>::gasViscosity<Evaluation>(co2Tables, temp, pressure, true).value();

    GpuBufCo2Tables gpuBufCo2Table = Opm::gpuistl::move_to_gpu<double, std::vector<double>, GpuB>(co2Tables);
    GpuViewCO2Tables gpuViewCo2Table = Opm::gpuistl::make_view<double, GpuB, GpuV>(gpuBufCo2Table);

    // Allocate memory for the result on the GPU
    double* resultOnGpu = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&resultOnGpu, sizeof(double)));

    // Allocate GPU memory for the Evaluation inputs
    Evaluation* gpuTemp = nullptr;
    Evaluation* gpuPressure = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuTemp, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuTemp, &temp, sizeof(Evaluation), hipMemcpyHostToDevice));
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuPressure, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuPressure, &pressure, sizeof(Evaluation), hipMemcpyHostToDevice));

    gasDiffCoeff<<<1,1>>>(gpuViewCo2Table, gpuTemp, gpuPressure, resultOnGpu);

    // Check for any errors in kernel launch
    OPM_GPU_SAFE_CALL(hipPeekAtLastError());
    OPM_GPU_SAFE_CALL(hipDeviceSynchronize());

    // Retrieve the result from the GPU to the CPU
    double resultOnCpu = 0.0;
    OPM_GPU_SAFE_CALL(hipMemcpy(&resultOnCpu, resultOnGpu, sizeof(double), hipMemcpyDeviceToHost));

    // Free allocated GPU memory
    OPM_GPU_SAFE_CALL(hipFree(resultOnGpu));
    OPM_GPU_SAFE_CALL(hipFree(gpuTemp));
    OPM_GPU_SAFE_CALL(hipFree(gpuPressure));

    // Verify that the CPU and GPU results match within a reasonable tolerance
    const double tolerance = 1e-6; // Tolerance for floating-point comparison
    BOOST_CHECK(std::fabs(resultOnCpu - enthalpyReference) < tolerance);
}

namespace {

// Kernel to use a BrineDynamic object on a GPU
__global__ void pvtInternalEnergy(GpuViewCo2Pvt gpuViewCo2Pvt, Evaluation* temp, Evaluation* pressure, double* result) {
    *result = gpuViewCo2Pvt.internalEnergy<Evaluation>(1, *temp, *pressure, Evaluation(0.4), Evaluation(0.0)).value();
}

bool compareSignificantDigits(double a, double b, int significantDigits) {
    // Handle the case where both values are exactly zero
    if (a == b) {
        return true;
    }

    // Calculate the relative error
    double relativeError = std::abs(a - b) / std::max(std::abs(a), std::abs(b));

    // Compute the number of matching digits
    double digitsMatched = -std::log10(relativeError);

    // Return true if they match the required number of significant digits
    return digitsMatched >= significantDigits;
}

} // END EMPTY NAMESPACE

// Test case evaluating pvt values for BrineDynamic on a GPU and CPU
BOOST_AUTO_TEST_CASE(TestCo2GasPvt) {
    Evaluation temp(290.5); // [K]
    Evaluation pressure(200000.0); // [Pa]
    std::vector<double> salinities = {0.2, 0.3, 0.4};
    // make a nonstatic version of the CPU CO2tables
    Opm::CO2Tables<double, std::vector<double>> co2Tables;

    CpuCo2Pvt cpuCo2Pvt(salinities);
    double internalEnergyReference = cpuCo2Pvt.internalEnergy<Evaluation>(1, temp, pressure, Evaluation(0.4), Evaluation(0.0)).value();

    const GpuBufCo2Pvt gpuBufCo2Pvt = Opm::gpuistl::move_to_gpu<double, GpuBufCo2Tables, GpuB>(cpuCo2Pvt);
    const auto brineReferenceDensityCPUCopy = gpuBufCo2Pvt.getBrineReferenceDensity().asStdVector();
    const GpuViewCo2Pvt gpuViewCo2Pvt = Opm::gpuistl::make_view<double, GpuBufCo2Tables, GpuViewCO2Tables, GpuB, GpuV>(gpuBufCo2Pvt);

    // Allocate memory for the result on the GPU
    double* resultOnGpu = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&resultOnGpu, sizeof(double)));

    // Allocate GPU memory for the Evaluation inputs
    Evaluation* gpuTemp = nullptr;
    Evaluation* gpuPressure = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuTemp, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuTemp, &temp, sizeof(Evaluation), hipMemcpyHostToDevice));
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuPressure, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuPressure, &pressure, sizeof(Evaluation), hipMemcpyHostToDevice));

    pvtInternalEnergy<<<1,1>>>(gpuViewCo2Pvt, gpuTemp, gpuPressure, resultOnGpu);

    // Check for any errors in kernel launch
    OPM_GPU_SAFE_CALL(hipPeekAtLastError());
    OPM_GPU_SAFE_CALL(hipDeviceSynchronize());

    // Retrieve the result from the GPU to the CPU
    double resultOnCpu = 0.0;
    OPM_GPU_SAFE_CALL(hipMemcpy(&resultOnCpu, resultOnGpu, sizeof(double), hipMemcpyDeviceToHost));

    // Free allocated GPU memory
    OPM_GPU_SAFE_CALL(hipFree(resultOnGpu));
    OPM_GPU_SAFE_CALL(hipFree(gpuTemp));
    OPM_GPU_SAFE_CALL(hipFree(gpuPressure));

    // Verify that the CPU and GPU results match within a reasonable tolerance
    const double tolerance = 1e-6; // Tolerance for floating-point comparison
    printf("%lf %lf\n", resultOnCpu, internalEnergyReference);
    // BOOST_CHECK(std::fabs(resultOnCpu - internalEnergyReference) < tolerance);
    BOOST_CHECK(compareSignificantDigits(resultOnCpu, internalEnergyReference, 6));
}


namespace {

// Kernel to use a BrineDynamic object on a GPU
__global__ void brineCo2PvtInternalEnergy(GpuViewBrineCo2Pvt gpuViewBrineCo2Pvt, Evaluation* temp, Evaluation* pressure, Evaluation* rs, Evaluation saltConcentration, double* result) {
    *result = gpuViewBrineCo2Pvt.internalEnergy<Evaluation>(1, *temp, *pressure, rs, saltConcentration).value();
}

} // END EMPTY NAMESPACE

// Test case evaluating pvt values for BrineDynamic on a GPU and CPU
BOOST_AUTO_TEST_CASE(TestBrineCo2Pvt) {
    Evaluation temp(290.5); // [K]
    Evaluation pressure(200000.0); // [Pa]
    Evaluation rs(0.3);
    Evaluation saltConcentration(0.1);

    std::vector<double> salinities = {0.2, 0.3, 0.4};
    // make a nonstatic version of the CPU CO2tables
    Opm::CO2Tables<double, std::vector<double>> co2Tables;

    CpuBrineCo2Pvt cpuBrineCo2Pvt(salinities);
    double internalEnergyReference = cpuBrineCo2Pvt.internalEnergy<Evaluation>(1, temp, pressure, rs, saltConcentration).value();

    const GpuViewBrineCo2Pvt gpuBufBrineCo2Pvt = Opm::gpuistl::move_to_gpu<double, GpuBufCo2Tables, GpuB>(cpuBrineCo2Pvt);
    const GpuViewBrineCo2Pvt gpuViewBrineCo2Pvt = Opm::gpuistl::make_view<double, GpuBufCo2Tables, GpuViewCO2Tables, GpuB, GpuV>(gpuBufCo2Pvt);

    // Allocate memory for the result on the GPU
    double* resultOnGpu = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&resultOnGpu, sizeof(double)));

    // Allocate GPU memory for the Evaluation inputs
    Evaluation* gpuRs = nullptr;
    Evaluation* gpuSaltConcentration = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuRs, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuRs, &rs, sizeof(Evaluation), hipMemcpyHostToDevice));
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuSaltConcentration, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuSaltConcentration, &saltConcentration, sizeof(Evaluation), hipMemcpyHostToDevice));
    Evaluation* gpuTemp = nullptr;
    Evaluation* gpuPressure = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuTemp, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuTemp, &temp, sizeof(Evaluation), hipMemcpyHostToDevice));
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuPressure, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuPressure, &pressure, sizeof(Evaluation), hipMemcpyHostToDevice));

    brineCo2PvtInternalEnergy<<<1,1>>>(gpuViewBrineCo2Pvt, gpuTemp, gpuPressure, gpuRs, gpuSaltConcentration, resultOnGpu);

    // Check for any errors in kernel launch
    OPM_GPU_SAFE_CALL(hipPeekAtLastError());
    OPM_GPU_SAFE_CALL(hipDeviceSynchronize());

    // Retrieve the result from the GPU to the CPU
    double resultOnCpu = 0.0;
    OPM_GPU_SAFE_CALL(hipMemcpy(&resultOnCpu, resultOnGpu, sizeof(double), hipMemcpyDeviceToHost));

    // Free allocated GPU memory
    OPM_GPU_SAFE_CALL(hipFree(resultOnGpu));
    OPM_GPU_SAFE_CALL(hipFree(gpuTemp));
    OPM_GPU_SAFE_CALL(hipFree(gpuPressure));

    // Verify that the CPU and GPU results match within a reasonable tolerance
    const double tolerance = 1e-6; // Tolerance for floating-point comparison
    printf("%lf %lf\n", resultOnCpu, internalEnergyReference);
    // BOOST_CHECK(std::fabs(resultOnCpu - internalEnergyReference) < tolerance);
    BOOST_CHECK(compareSignificantDigits(resultOnCpu, internalEnergyReference, 6));
}
