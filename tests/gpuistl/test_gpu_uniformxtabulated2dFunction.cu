#include "hip/hip_runtime.h"
/*
  Copyright 2024 SINTEF AS
  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <config.h>

#define BOOST_TEST_MODULE TestGpuUniformXTabulated2DFunction

#include <boost/test/unit_test.hpp>
#include <opm/material/densead/Evaluation.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpu_safe_call.hpp>
#include <opm/material/common/UniformXTabulated2DFunction.hpp>
#include <opm/simulators/linalg/gpuistl/GpuBuffer.hpp>
#include <opm/simulators/linalg/gpuistl/GpuView.hpp>
#include <hip/hip_runtime.h>

namespace{
__global__ void instansiate_ad_object(Opm::DenseAd::Evaluation<float, 3>* adObj, double value){
    *adObj = Opm::DenseAd::Evaluation<float, 3>(value, 0);
}

//TODO add more comprenehsive AD tests

} // END EMPTY NAMESPACE


BOOST_AUTO_TEST_CASE(TestInstansiateADObject)
{
    using CpuTable = Opm::UniformXTabulated2DFunction<double>;
    using GpuTable = Opm::UniformXTabulated2DFunction<double, Opm::gpuistl::GpuBuffer>;

    Opm::UniformXTabulated2DFunction<double> cpuTable;
    cpuTable.appendXPos(0.1);
    cpuTable.appendXPos(0.15);
    cpuTable.appendXPos(0.5);
    cpuTable.appendXPos(0.9);
    cpuTable.appendSamplePoint(0, 0.3, 2.0);
    cpuTable.appendSamplePoint(1, 0.5, 5.0);
    cpuTable.appendSamplePoint(2, 0.8, 20.0);
    cpuTable.appendSamplePoint(3, 0.9, 3.0);

    GpuTable gpuTable = Opm::gpuistl::moveToGpu<double, CpuTable, GpuTable, Opm::gpuistl::GpuBuffer>(cpuTable);

    // using Evaluation = Opm::DenseAd::Evaluation<float, 3>;
    // double testValue = 123.456;
    // Evaluation cpuMadeAd = Evaluation(testValue, 0);

    // Evaluation gpuMadeAd[1]; // allocate space for one more AD object on the CPU
    // Evaluation *d_ad;

    // // allocate space on GPU, run kernel, and move results back to the CPU
    // OPM_GPU_SAFE_CALL(hipMalloc(&d_ad, sizeof(Evaluation)));
    // instansiate_ad_object<<<1,1>>>(d_ad, testValue);
    // OPM_GPU_SAFE_CALL(hipDeviceSynchronize());
    // OPM_GPU_SAFE_CALL(hipMemcpy(&gpuMadeAd, d_ad, sizeof(Evaluation), hipMemcpyDeviceToHost));
    // OPM_GPU_SAFE_CALL(hipFree(d_ad));

    // // Check that the object made in a GPU kernel is equivalent to that made on the CPU
    // BOOST_CHECK(cpuMadeAd == gpuMadeAd[0]);
}
