#include "hip/hip_runtime.h"
/*
  Copyright 2025 SINTEF AS
  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#if 0
#include "config.h"

#include <opm/simulators/flow/Main.hpp>
#include <opm/simulators/flow/FlowMain.hpp>
//#define BOOST_TEST_MODULE TestFlowProblemGpu

//#include <boost/test/unit_test.hpp>

#include <opm/material/densead/Evaluation.hpp>
#include <opm/material/fluidmatrixinteractions/EclMaterialLawManagerSimple.hpp>

#include <opm/models/blackoil/blackoilmodel.hh>
#include <opm/models/discretization/common/tpfalinearizer.hh>
#include <opm/models/utils/simulator.hh>

#include <opm/simulators/utils/moduleVersion.hpp>
#include <opm/simulators/flow/FlowProblemBlackoilGpu.hpp>
#include <opm/simulators/flow/FlowProblemBlackoil.hpp>
#include <opm/simulators/flow/FlowProblemBlackoilProperties.hpp>

#include <opm/simulators/flow/BlackoilModelParameters.hpp>
#include <opm/simulators/flow/FlowGenericVanguard.hpp>
#include <opm/simulators/flow/FlowProblemBlackoil.hpp>
#include <opm/simulators/flow/FlowProblemBlackoilProperties.hpp>
#include <opm/simulators/flow/equil/EquilibrationHelpers.hpp>
#include <opm/simulators/linalg/parallelbicgstabbackend.hh>
#include <opm/simulators/linalg/gpuistl/GpuBuffer.hpp>
#include <opm/simulators/linalg/gpuistl/DualBuffer.hpp>
#include <opm/simulators/linalg/gpuistl/GpuView.hpp>
#include <opm/simulators/linalg/gpuistl/gpu_smart_pointer.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpu_safe_call.hpp>
#include <opm/simulators/wells/BlackoilWellModel.hpp>

#include <utility>

#include <hip/hip_runtime.h>


#include <fmt/core.h>

/*
Functionality requested for the blackoil flow problem on gpu:
[X] indicates that the functionality is added and verified with unit test
[-] indicates that the functionality does not seem to be used in spe11
[ ] indicates that the functionality is not added yet

[X] - problem.model().linearizer().getLinearizationType()
[X] - problem.satnumRegionIndex(globalSpaceIdx)
[X] - problem.materialLawParams(globalSpaceIdx)
[X] - problem.rockCompressibility(globalSpaceIdx)
[X] - problem.rockReferencePressure(globalSpaceIdx)
[X] - problem.porosity(globalSpaceIdx, timeIdx)
[-] - problem.maxOilVaporizationFactor(timeIdx, globalSpaceIdx)
[-] - problem.maxGasDissolutionFactor(timeIdx, globalSpaceIdx)
[-] - problem.maxOilSaturation(globalSpaceIdx)
[-] - problem.template rockCompPoroMultiplier<Evaluation>(*this, globalSpaceIdx)
[X] - problem.updateRelperms(mobility_, dirMob_, fluidState_, globalSpaceIdx)
[X] - problem.template rockCompTransMultiplier<Evaluation>(*this, globalSpaceIdx)

*/

#include <opm/material/fluidsystems/BlackOilFluidSystem.hpp>
#include <opm/material/fluidsystems/BlackOilFluidSystemNonStatic.hpp>
#include <opm/material/fluidstates/BlackOilFluidState.hpp>

#include <opm/input/eclipse/Parser/Parser.hpp>
#include <opm/input/eclipse/Deck/Deck.hpp>
#include <opm/input/eclipse/EclipseState/EclipseState.hpp>
#include <opm/input/eclipse/Python/Python.hpp>
#include <opm/input/eclipse/Schedule/Schedule.hpp>

static constexpr const char* deckString1 =
"-- =============== RUNSPEC\n"
"RUNSPEC\n"
"DIMENS\n"
"3 3 3 /\n"
"EQLDIMS\n"
"/\n"
"TABDIMS\n"
"/\n"
"WATER\n"
"GAS\n"
"CO2STORE\n"
"METRIC\n"
"-- =============== GRID\n"
"GRID\n"
"GRIDFILE\n"
"0 0 /\n"
"DX\n"
"27*1 /\n"
"DY\n"
"27*1 /\n"
"DZ\n"
"27*1 /\n"
"TOPS\n"
"9*0 /\n"
"PERMX\n"
"27*1013.25 /\n"
"PORO\n"
"27*0.25 /\n"
"COPY\n"
"PERMX PERMY /\n"
"PERMX PERMZ /\n"
"/\n"
"-- =============== PROPS\n"
"PROPS\n"
"SGWFN\n"
"0.000000E+00 0.000000E+00 1.000000E+00 3.060000E-02\n"
"1.000000E+00 1.000000E+00 0.000000E+00 3.060000E-01 /\n"
"-- =============== SOLUTION\n"
"SOLUTION\n"
"RPTRST\n"
"'BASIC=0' /\n"
"EQUIL\n"
"0 300 100 0 0 0 1 1 0 /\n"
"-- =============== SCHEDULE\n"
"SCHEDULE\n"
"RPTRST\n"
"'BASIC=0' /\n"
"TSTEP\n"
"1 /";


namespace Opm {
  namespace Properties {
      namespace TTag {
          struct FlowSimpleProblem {
              using InheritsFrom = std::tuple<FlowProblem>;
          };
      }

      // Indices for two-phase gas-water.
      template<class TypeTag>
      struct Indices<TypeTag, TTag::FlowSimpleProblem>
      {
      private:
          // it is unfortunately not possible to simply use 'TypeTag' here because this leads
          // to cyclic definitions of some properties. if this happens the compiler error
          // messages unfortunately are *really* confusing and not really helpful.
          using BaseTypeTag = TTag::FlowProblem;
          using FluidSystem = GetPropType<BaseTypeTag, Properties::FluidSystem>;

      public:
          using type = BlackOilTwoPhaseIndices<getPropValue<TypeTag, Properties::EnableSolvent>(),
                                              getPropValue<TypeTag, Properties::EnableExtbo>(),
                                              getPropValue<TypeTag, Properties::EnablePolymer>(),
                                              getPropValue<TypeTag, Properties::EnableEnergy>(),
                                              getPropValue<TypeTag, Properties::EnableFoam>(),
                                              getPropValue<TypeTag, Properties::EnableBrine>(),
                                              /*PVOffset=*/0,
                                              /*disabledCompIdx=*/FluidSystem::oilCompIdx,
                                              getPropValue<TypeTag, Properties::EnableMICP>()>;
      };

      // SPE11C requires thermal/energy
      // READD THIS ARGUMENT
      // template<class TypeTag>
      // struct EnableEnergy<TypeTag, TTag::FlowSimpleProblem> {
      //     static constexpr bool value = true;
      // };

      // SPE11C requires dispersion
      template<class TypeTag>
      struct EnableDispersion<TypeTag, TTag::FlowSimpleProblem> {
          static constexpr bool value = true;
      };

      // Use the simple material law.
      template<class TypeTag>
      struct MaterialLaw<TypeTag, TTag::FlowSimpleProblem>
      {
      private:
          using Scalar = GetPropType<TypeTag, Properties::Scalar>;
          using FluidSystem = GetPropType<TypeTag, Properties::FluidSystem>;

          using Traits = ThreePhaseMaterialTraits<Scalar,
                                                  /*wettingPhaseIdx=*/FluidSystem::waterPhaseIdx,
                                                  /*nonWettingPhaseIdx=*/FluidSystem::oilPhaseIdx,
                                                  /*gasPhaseIdx=*/FluidSystem::gasPhaseIdx>;
      public:
          using EclMaterialLawManager = ::Opm::EclMaterialLawManagerSimple<Traits>;
          using type = typename EclMaterialLawManager::MaterialLaw;
      };

      // Use the TPFA linearizer.
      template<class TypeTag>
      struct Linearizer<TypeTag, TTag::FlowSimpleProblem> { using type = TpfaLinearizer<TypeTag>; };

      template<class TypeTag>
      struct LocalResidual<TypeTag, TTag::FlowSimpleProblem> { using type = BlackOilLocalResidualTPFA<TypeTag>; };

      // Diffusion.
      template<class TypeTag>
      struct EnableDiffusion<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };

      template<class TypeTag>
      struct EnableDisgasInWater<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };

      template<class TypeTag>
      struct EnableVapwat<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };

  };

}

// namespace Opm {
//   namespace Properties {
//       namespace TTag {
//           struct FlowSimpleProblem {
//               using InheritsFrom = std::tuple<FlowProblem>;
//           };
//       }

//       // Indices for two-phase gas-water.
//       template<class TypeTag>
//       struct Indices<TypeTag, TTag::FlowSimpleProblem>
//       {
//       private:
//           // it is unfortunately not possible to simply use 'TypeTag' here because this leads
//           // to cyclic definitions of some properties. if this happens the compiler error
//           // messages unfortunately are *really* confusing and not really helpful.
//           using BaseTypeTag = TTag::FlowProblem;
//           using FluidSystem = GetPropType<BaseTypeTag, Properties::FluidSystem>;

//       public:
//           using type = BlackOilTwoPhaseIndices<getPropValue<TypeTag, Properties::EnableSolvent>(),
//                                               getPropValue<TypeTag, Properties::EnableExtbo>(),
//                                               getPropValue<TypeTag, Properties::EnablePolymer>(),
//                                               getPropValue<TypeTag, Properties::EnableEnergy>(),
//                                               getPropValue<TypeTag, Properties::EnableFoam>(),
//                                               getPropValue<TypeTag, Properties::EnableBrine>(),
//                                               /*PVOffset=*/0,
//                                               /*disabledCompIdx=*/FluidSystem::oilCompIdx,
//                                               getPropValue<TypeTag, Properties::EnableMICP>()>;
//       };

//       // SPE11C requires thermal/energy
//       // template<class TypeTag>
//       // struct EnableEnergy<TypeTag, TTag::FlowSimpleProblem> {
//       //     static constexpr bool value = true;
//       // };

//       // SPE11C requires dispersion
//       template<class TypeTag>
//       struct EnableDispersion<TypeTag, TTag::FlowSimpleProblem> {
//           static constexpr bool value = true;
//       };

//       // Use the simple material law.
//       template<class TypeTag>
//       struct MaterialLaw<TypeTag, TTag::FlowSimpleProblem>
//       {
//       private:
//           using Scalar = GetPropType<TypeTag, Properties::Scalar>;
//           using FluidSystem = GetPropType<TypeTag, Properties::FluidSystem>;
//       public:
//           using Traits = ThreePhaseMaterialTraits<Scalar,
//                                                   /*wettingPhaseIdx=*/FluidSystem::waterPhaseIdx,
//                                                   /*nonWettingPhaseIdx=*/FluidSystem::oilPhaseIdx,
//                                                   /*gasPhaseIdx=*/FluidSystem::gasPhaseIdx>;
//           using EclMaterialLawManager = ::Opm::EclMaterialLawManagerSimple<Traits>;
//           using type = typename EclMaterialLawManager::MaterialLaw;
//       };

//       // Use the TPFA linearizer.
//       template<class TypeTag>
//       struct Linearizer<TypeTag, TTag::FlowSimpleProblem> { using type = TpfaLinearizer<TypeTag>; };

//       template<class TypeTag>
//       struct LocalResidual<TypeTag, TTag::FlowSimpleProblem> { using type = BlackOilLocalResidualTPFA<TypeTag>; };

//       // Diffusion.
//       template<class TypeTag>
//       struct EnableDiffusion<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };

//       template<class TypeTag>
//       struct EnableDisgasInWater<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };

//       template<class TypeTag>
//       struct EnableVapwat<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };
//       // template<class TypeTag>
//       // struct PrimaryVariables<TypeTag, TTag::FlowSimpleProblem> { using type = BlackOilPrimaryVariables<TypeTag, Opm::gpuistl::dense::FieldVector>; };
//   };

// }


#include <iostream>
#include <type_traits>
#include <memory>
// #include <dune/common/mpihelper.hh>
#include <dune/common/parallel/mpihelper.hh>
#include <opm/models/utils/start.hh>

template<class ProblemView>
__global__ void satnumFromFlowProblemBlackoilGpu(ProblemView prob, unsigned short* res)
{
  *res = prob.satnumRegionIndex(0);
}

template<class ProblemView>
__global__ void linTypeFromFlowProblemBlackoilGpu(ProblemView prob, Opm::LinearizationType* res)
{
  *res = prob.model().linearizer().getLinearizationType();
}

template<class ProblemView>
__global__ void rockCompressibilityFromFlowProblemBlackoilGpu(ProblemView prob, double* res)
{
  *res = prob.rockCompressibility(0);
}

template<class ProblemView>
__global__ void porosityFromFlowProblemBlackoilGpu(ProblemView prob, double* res)
{
  *res = prob.porosity(0, 0);
}

template<class ProblemView>
__global__ void rockReferencePressureFromFlowProblemBlackoilGpu(ProblemView prob, double* res)
{
  *res = prob.rockReferencePressure(0);
}

template<class ProblemView>
__global__ void materialLawParamsCallable(ProblemView prob)
{
  auto matLawParams = prob.materialLawParams(0);
}

template<class DirMobPtr, class ProblemView, class MobArr, class FluidState>
__global__ void updateRelPermsFromFlowProblemBlackoilGpu(ProblemView prob, MobArr mob, FluidState fs)
{
  auto dirPtr = DirMobPtr(); // produces nullptr, this value is not used in the function, but should match signature
  prob.updateRelperms(mob, dirPtr, fs, 0);
}



//BOOST_AUTO_TEST_CASE(TestInstantiateGpuFlowProblem)
//{
int main(int realArgc, char** realArgv) {
  using TypeTag = Opm::Properties::TTag::FlowSimpleProblem;
  // BOOST_CHECK(true);
  std::vector<std::string> args = {"test_gpuflowproblem", "very_simple_deck.DATA", ""};
  std::vector<char*> argv;
  for (auto& arg : args) {
    argv.push_back(static_cast<char*>(arg.data()));
  }

  // Check if the file specified in args[1] exists
  {
    std::ifstream file(args[1]);
    if (!file.good()) {
      throw std::runtime_error("File not found: " + args[1]);
    }
  }

  fmt::print("New version");
  fmt::print("realArgc: {}", realArgc);
  fmt::print("realArgv: {}", realArgv[0]);
  // BOOST_CHECK(true);
  //auto mainObject = Opm::Main(argv.size(), static_cast<char**>(argv.data()));
  auto mainObject = Opm::Main(realArgc, realArgv);
  // BOOST_CHECK(true);

  // char* argv[] = {const_cast<char*>("very_simple_deck.DATA")};
  // auto mainObject = Opm::Main(1, argv);
  mainObject.runStatic<TypeTag>();
  // mainObject.setupVanguard();
  // BOOST_CHECK(true);
  // auto flowMainObject = Opm::FlowMain<TypeTag>(argv.size(), argv.data(), false, false);
  // BOOST_CHECK(true);
  // // flowMainObject.runStatic<TypeTag>();
  // auto sim1 = flowMainObject.get_simulator();
  // BOOST_CHECK(true);
  // auto& sim = sim1->getSimulator();
  // BOOST_CHECK(true);
  // auto& problem = sim.problem();
  // auto& matLawParams = problem.materialLawParams(0);
  // auto& gasWaterParams = matLawParams.gasWaterParams();
  // gasWaterParams.printme();


//   using TypeTag = Opm::Properties::TTag::FlowSimpleProblem;
//   // FIXTURE FROM TEST EQUIL
//   int argc1 = boost::unit_test::framework::master_test_suite().argc;
//   char** argv1 = boost::unit_test::framework::master_test_suite().argv;

// #if HAVE_DUNE_FEM
//   Dune::Fem::MPIManager::initialize(argc1, argv1);
// #else
//   Dune::MPIHelper::instance(argc1, argv1);
// #endif

//   using namespace Opm;
//   FlowGenericVanguard::setCommunication(std::make_unique<Opm::Parallel::Communication>());
//   Opm::ThreadManager::registerParameters();
//   BlackoilModelParameters<double>::registerParameters();
//   AdaptiveTimeStepping<TypeTag>::registerParameters();
//   Parameters::Register<Parameters::EnableTerminalOutput>("Dummy added for the well model to compile.");
//   registerAllParameters_<TypeTag>();

//   // END OF FIXTURE FROM TEST EQUIL

//   using Simulator = Opm::GetPropType<TypeTag, Opm::Properties::Simulator>;

//   // TODO: will this actually refer to the very_simple_deck.DATA inside the gpuistl folder,
//   // TODO: do we need to keep track of the path since it can be hipified?
//   const char* filename = "very_simple_deck.DATA";
//   const auto filenameArg = std::string {"--ecl-deck-file-name="} + filename;

//   const char* argv2[] = {
//       "test_gpuflowproblem",
//       filenameArg.c_str(),
//       "--check-satfunc-consistency=false",
//   };

//   Opm::setupParameters_<TypeTag>(/*argc=*/sizeof(argv2)/sizeof(argv2[0]), argv2, /*registerParams=*/false);

//   Opm::FlowGenericVanguard::readDeck(filename);

//   auto sim = std::make_unique<Simulator>();

//   using ThreePhaseTraits = typename GetPropType<TypeTag, Properties::MaterialLaw>::Traits;

//   // using ThreePhaseParams = TypeTag::MaterialLaw::EclMaterialLawManager::MaterialLawParams;
//   // using ThreePhaseParams = typename GetPropType<TypeTag, Properties::MaterialLaw>::EclMaterialLawManager::MaterialLawParams;
//   using ThreePhaseParams = typename ::Opm::EclMaterialLawManagerSimple<ThreePhaseTraits>::MaterialLawParams;
//   using CpuGasWaterTwoPhaseLaw = ThreePhaseParams::GasWaterParams;

//   enum { waterPhaseIdx = ThreePhaseTraits::wettingPhaseIdx };
//   enum { oilPhaseIdx = ThreePhaseTraits::nonWettingPhaseIdx };
//   enum { gasPhaseIdx = ThreePhaseTraits::gasPhaseIdx };
//   enum { numPhases = ThreePhaseTraits::numPhases };
//   using GasWaterTraits = TwoPhaseMaterialTraits<double, waterPhaseIdx, gasPhaseIdx>;

//   using GPUBufferInterpolation = Opm::PiecewiseLinearTwoPhaseMaterialParams<GasWaterTraits, Opm::gpuistl::GpuBuffer<double>>;
//   using GPUViewInterpolation = Opm::PiecewiseLinearTwoPhaseMaterialParams<GasWaterTraits, Opm::gpuistl::GpuView<double>>;

//   auto problemGpuBuf = Opm::gpuistl::copy_to_gpu<double, Opm::gpuistl::GpuBuffer, Opm::gpuistl::DualBuffer, TypeTag, TypeTag>(sim->problem());
  // auto problemGpuView = Opm::gpuistl::make_view<Opm::gpuistl::GpuView, Opm::gpuistl::ValueAsPointer>(problemGpuBuf);

  // unsigned short satNumOnCpu;
  // unsigned short* satNumOnGpu;
  // std::ignore = hipMalloc(&satNumOnGpu, sizeof(unsigned short));
  // satnumFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, satNumOnGpu);
  // std::ignore = hipMemcpy(&satNumOnCpu, satNumOnGpu, sizeof(unsigned short), hipMemcpyDeviceToHost);
  // BOOST_CHECK_EQUAL(satNumOnCpu, sim->problem().satnumRegionIndex(0));
  // std::ignore = hipFree(satNumOnGpu);

  // Opm::LinearizationType linTypeOnCpu;
  // Opm::LinearizationType* linTypeOnGpu;
  // std::ignore = hipMalloc(&linTypeOnGpu, sizeof(Opm::LinearizationType));
  // linTypeFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, linTypeOnGpu);
  // std::ignore = hipMemcpy(&linTypeOnCpu, linTypeOnGpu, sizeof(Opm::LinearizationType), hipMemcpyDeviceToHost);
  // auto linTypeFromCPUSimulator = sim->problem().model().linearizer().getLinearizationType();
  // BOOST_CHECK_EQUAL(linTypeOnCpu.type, linTypeFromCPUSimulator.type);
  // std::ignore = hipFree(linTypeOnGpu);

  // double rocmCompressibilityOnCpu;
  // double* rockCompressibilityOnGpu;
  // std::ignore = hipMalloc(&rockCompressibilityOnGpu, sizeof(double));
  // rockCompressibilityFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, rockCompressibilityOnGpu);
  // std::ignore = hipMemcpy(&rocmCompressibilityOnCpu, rockCompressibilityOnGpu, sizeof(double), hipMemcpyDeviceToHost);
  // BOOST_CHECK_EQUAL(rocmCompressibilityOnCpu, sim->problem().rockCompressibility(0));
  // std::ignore = hipFree(rockCompressibilityOnGpu);

  // double porosityOnCpu;
  // double* porosityOnGpu;
  // std::ignore = hipMalloc(&porosityOnGpu, sizeof(double));
  // porosityFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, porosityOnGpu);
  // std::ignore = hipMemcpy(&porosityOnCpu, porosityOnGpu, sizeof(double), hipMemcpyDeviceToHost);
  // BOOST_CHECK_EQUAL(porosityOnCpu, sim->problem().porosity(0, 0));
  // std::ignore = hipFree(porosityOnGpu);

  // double referencePressureOnCpu;
  // double* referencePressureOnGpu;
  // std::ignore = hipMalloc(&referencePressureOnGpu, sizeof(double));
  // rockReferencePressureFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, referencePressureOnGpu);
  // std::ignore = hipMemcpy(&referencePressureOnCpu, referencePressureOnGpu, sizeof(double), hipMemcpyDeviceToHost);
  // BOOST_CHECK_EQUAL(referencePressureOnCpu, sim->problem().rockReferencePressure(0));
  // std::ignore = hipFree(referencePressureOnGpu);

  // materialLawParamsCallable<<<1, 1>>>(problemGpuView);

  // using FluidSystem = Opm::BlackOilFluidSystem<double>;
  // using Evaluation = Opm::DenseAd::Evaluation<double,2>;
  // using Scalar = double;
  // // using DirectionalMobilityPtr = Utility::CopyablePtr<DirectionalMobility<TypeTag, Evaluation>>;
  // using DirectionalMobilityPtr = Utility::CopyablePtr<DirectionalMobility<TypeTag>>;
  
  // // Create the fluid system
  // Opm::Parser parser;
  // auto deck = parser.parseString(deckString1);
  // auto python = std::make_shared<Opm::Python>();
  // Opm::EclipseState eclState(deck);
  // Opm::Schedule schedule(deck, eclState, python);

  // FluidSystem::initFromState(eclState, schedule);
  // auto& dynamicFluidSystem = FluidSystem::getNonStaticInstance();
  // auto dynamicGpuFluidSystemBuffer = ::Opm::gpuistl::copy_to_gpu<::Opm::gpuistl::GpuBuffer, double>(dynamicFluidSystem);
  // auto dynamicGpuFluidSystemView = ::Opm::gpuistl::make_view<::Opm::gpuistl::GpuView, ::Opm::gpuistl::ValueAsPointer>(dynamicGpuFluidSystemBuffer);
  // auto gpufluidstate = BlackOilFluidState<double, decltype(dynamicGpuFluidSystemView)>(dynamicGpuFluidSystemView);
  // // Create MobArr
  // double testValue = 0.5;
  // // Create an array of Evaluations on CPU
  // using MobArr = std::array<Evaluation, 2>;
  // MobArr cpuMobArray;
  // cpuMobArray[0] = Evaluation(testValue, 0);
  // cpuMobArray[1] = Evaluation(testValue, 1);
  
  // // Copy to GPU
  // MobArr* d_mobArray;
  // OPM_GPU_SAFE_CALL(hipMalloc(&d_mobArray, sizeof(MobArr)));
  // OPM_GPU_SAFE_CALL(hipMemcpy(d_mobArray, &cpuMobArray, sizeof(MobArr), hipMemcpyHostToDevice));
  
  // updateRelPermsFromFlowProblemBlackoilGpu<DirectionalMobilityPtr><<<1, 1>>>(problemGpuView, *d_mobArray, gpufluidstate);
}
#else 
/*
  Copyright 2024, SINTEF AS

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
// #include "config.h"
#define HAS_ATTRIBUTE_UNUSED 1
#define HAS_ATTRIBUTE_DEPRECATED 1
#define HAS_ATTRIBUTE_DEPRECATED_MSG 1
/* #undef HAVE_ARRAY */
/* #undef HAVE_BOOST_MAKE_SHARED_HPP */
/* #undef HAVE_BOOST_SHARED_PTR_HPP */
#define HAVE_DUNE_BOOST 1
/* #undef HAVE_GMP */
/* #undef HAVE_MAKE_SHARED */
#define HAVE_MPI 1
/* #undef HAVE_NULLPTR */
/* #undef HAVE_STATIC_ASSERT */
/* #undef HAVE_SHARED_PTR */
#define MPI_2 1
/* #undef SHARED_PTR_HEADER */
/* #undef SHARED_PTR_NAMESPACE */
/* #undef HAVE_TYPE_TRAITS */
/* #undef HAVE_TR1_TUPLE */
/* #undef HAVE_TUPLE */
#define HAVE_CXA_DEMANGLE 1
/* #undef HAVE_BOOST_FUSION */
/* #undef HAVE_MEM_USAGE_T_EXPANSIONS */
/* #undef HAVE_PARDISO */
#define HAVE_BOOST 1
/* #undef HAVE_PARMETIS */
#define HAVE_SUPERLU 1
#define HAVE_SUITESPARSE_UMFPACK 1
#define SUPERLU_INT_TYPE int
#define HAVE_OPENMP 1
#define HAVE_VALGRIND 1
/* #undef HAVE_FINAL */
#define HAVE_ECL_INPUT 1
#define HAVE_FNMATCH_H 1
/* #undef HAVE_ALGLIB */
/* #undef HAVE_UG */
/* #undef HAVE_DUNE_FEM */
/* #undef HAVE_GRIDTYPE */
/* #undef HAVE_GRAPE */
/* #undef HAVE_PSURFACE */
/* #undef HAVE_AMIRAMESH */
#define HAVE_ALBERTA 1
/* #undef HAVE_STDINT_H */
#define DUNE_GRID_EXPERIMENTAL_GRID_EXTENSIONS 1
#define DUNE_ALBERTA_VERSION 0x300
#define DUNE_GRID_VERSION_MAJOR 2
#define DUNE_GRID_VERSION_MINOR 9
#define DUNE_GRID_VERSION_REVISION 0
#define DUNE_GEOMETRY_VERSION_MAJOR 2
#define DUNE_GEOMETRY_VERSION_MINOR 9
#define DUNE_GEOMETRY_VERSION_REVISION 0
#define DUNE_COMMON_VERSION_MAJOR 2
#define DUNE_COMMON_VERSION_MINOR 9
#define DUNE_COMMON_VERSION_REVISION 1
#define HAVE_DUNE_ISTL 1
/* #undef HAVE_METIS */
/* #undef HAVE_PTSCOTCH */
#define IS_SCOTCH_METIS_HEADER 0
#define HAVE_ZOLTAN 1
#define HAVE_OPM_COMMON 1
#define HAVE_OPM_GRID 1
#define HAVE_PTHREAD 1
/* #undef HAVE_EWOMS */
/* #undef HAVE_PETSC */
/* #undef COMPILE_GPU_BRIDGE */
#define HAVE_CUDA 1
#define HAVE_OPENCL 1
#define HAVE_OPENCL_HPP 1
#define HAVE_AMGCL 1
/* #undef HAVE_AMGX */
/* #undef HAVE_VEXCL */
#define HAVE_ROCALUTION 1
#define HAVE_ROCSPARSE 1
#define HAVE_SUITESPARSE_UMFPACK_H 1
#define HAVE_DUNE_COMMON 1
/* #undef DUNE_ISTL_WITH_CHECKING */
#define DUNE_ISTL_VERSION_MAJOR 2
#define DUNE_ISTL_VERSION_MINOR 9
#define DUNE_ISTL_VERSION_REVISION 0
/* #undef HAVE_DAMARIS */
#define HAVE_HDF5 1
/* #undef HAVE_HYPRE */
#define USE_HIP 1
/* #undef USE_TRACY */
/* #undef FLOW_INSTANTIATE_FLOAT */
#define HAVE_FLOATING_POINT_FROM_CHARS 1
#define OPM_COMPILE_COMPONENTS_TEMPLATE_LIST 2,3,4,5,6,7
#define HAVE_DYNAMIC_BOOST_TEST 1


// #define BOOST_TEST_MODULE TestFlowSimple

// #include <boost/test/unit_test.hpp>

#include <opm/simulators/flow/Main.hpp>
#include <opm/material/fluidmatrixinteractions/EclMaterialLawManagerSimple.hpp>
#include <opm/models/discretization/common/tpfalinearizer.hh>
// do I need these?
#include <opm/simulators/flow/equil/EquilibrationHelpers.hpp>
#include <opm/simulators/flow/equil/InitStateEquil.hpp>

namespace Opm {
    namespace Properties {
        namespace TTag {
            struct FlowSimpleProblem {
                using InheritsFrom = std::tuple<FlowProblem>;
            };
        }

        // Indices for two-phase gas-water.
        template<class TypeTag>
        struct Indices<TypeTag, TTag::FlowSimpleProblem>
        {
        private:
            // it is unfortunately not possible to simply use 'TypeTag' here because this leads
            // to cyclic definitions of some properties. if this happens the compiler error
            // messages unfortunately are *really* confusing and not really helpful.
            using BaseTypeTag = TTag::FlowProblem;
            using FluidSystem = GetPropType<BaseTypeTag, Properties::FluidSystem>;

        public:
            using type = BlackOilTwoPhaseIndices<getPropValue<TypeTag, Properties::EnableSolvent>(),
                                                getPropValue<TypeTag, Properties::EnableExtbo>(),
                                                getPropValue<TypeTag, Properties::EnablePolymer>(),
                                                getPropValue<TypeTag, Properties::EnableEnergy>(),
                                                getPropValue<TypeTag, Properties::EnableFoam>(),
                                                getPropValue<TypeTag, Properties::EnableBrine>(),
                                                /*PVOffset=*/0,
                                                /*disabledCompIdx=*/FluidSystem::oilCompIdx,
                                                getPropValue<TypeTag, Properties::EnableMICP>()>;
        };

        // SPE11C requires thermal/energy
        // READD THIS ARGUMENT
        // template<class TypeTag>
        // struct EnableEnergy<TypeTag, TTag::FlowSimpleProblem> {
        //     static constexpr bool value = true;
        // };

        // SPE11C requires dispersion
        template<class TypeTag>
        struct EnableDispersion<TypeTag, TTag::FlowSimpleProblem> {
            static constexpr bool value = true;
        };

        // Use the simple material law.
        template<class TypeTag>
        struct MaterialLaw<TypeTag, TTag::FlowSimpleProblem>
        {
        private:
            using Scalar = GetPropType<TypeTag, Properties::Scalar>;
            using FluidSystem = GetPropType<TypeTag, Properties::FluidSystem>;

            using Traits = ThreePhaseMaterialTraits<Scalar,
                                                    /*wettingPhaseIdx=*/FluidSystem::waterPhaseIdx,
                                                    /*nonWettingPhaseIdx=*/FluidSystem::oilPhaseIdx,
                                                    /*gasPhaseIdx=*/FluidSystem::gasPhaseIdx>;
        public:
            using EclMaterialLawManager = ::Opm::EclMaterialLawManagerSimple<Traits>;
            using type = typename EclMaterialLawManager::MaterialLaw;
        };

        // Use the TPFA linearizer.
        template<class TypeTag>
        struct Linearizer<TypeTag, TTag::FlowSimpleProblem> { using type = TpfaLinearizer<TypeTag>; };

        template<class TypeTag>
        struct LocalResidual<TypeTag, TTag::FlowSimpleProblem> { using type = BlackOilLocalResidualTPFA<TypeTag>; };

        // Diffusion.
        template<class TypeTag>
        struct EnableDiffusion<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };

        template<class TypeTag>
        struct EnableDisgasInWater<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };

        template<class TypeTag>
        struct EnableVapwat<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };

    };

}
#ifndef BOOST_CHECK
#define BOOST_CHECK(x) std::cout << __LINE__ << std::endl;
#endif
// BOOST_AUTO_TEST_CASE(TestFlowSimple)
int main()
{

  using TypeTag = Opm::Properties::TTag::FlowSimpleProblem;
  BOOST_CHECK(true);
  std::vector<std::string> args = {"./../../../super_build_release/opm-simulators/bin/flow_simple", "very_simple_deck.DATA", ""};
  std::vector<char*> argv;
  for (auto& arg : args) {
    argv.push_back(static_cast<char*>(arg.data()));
  }

  // Check if the file specified in args[1] exists
  {
    std::ifstream file(args[1]);
    if (!file.good()) {
      throw std::runtime_error("File not found: " + args[1]);
    }
  }
  BOOST_CHECK(true);
  auto mainObject = Opm::Main(argv.size(), static_cast<char**>(argv.data()));
  BOOST_CHECK(true);
  
  // char* argv[] = {const_cast<char*>("very_simple_deck.DATA")};
  // auto mainObject = Opm::Main(1, argv);
  mainObject.runStatic<TypeTag>();
//    return Opm::start<TypeTag>(argc, argv);
}
#endif
