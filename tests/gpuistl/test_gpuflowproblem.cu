#include "hip/hip_runtime.h"
/*
  Copyright 2024 SINTEF AS
  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <config.h>

#define BOOST_TEST_MODULE TestFlowProblemGpu

#include <boost/test/unit_test.hpp>
#include <opm/material/densead/Evaluation.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpu_safe_call.hpp>
#include <opm/simulators/flow/FlowProblemBlackoilGpu.hpp>
#include <hip/hip_runtime.h>

// namespace Opm {
//   namespace Properties {
//       namespace TTag {
//           struct FlowSimpleProblem {
//               using InheritsFrom = std::tuple<FlowProblem>;
//           };
//       }

//       // Indices for two-phase gas-water.
//       template<class TypeTag>
//       struct Indices<TypeTag, TTag::FlowSimpleProblem>
//       {
//       private:
//           // it is unfortunately not possible to simply use 'TypeTag' here because this leads
//           // to cyclic definitions of some properties. if this happens the compiler error
//           // messages unfortunately are *really* confusing and not really helpful.
//           using BaseTypeTag = TTag::FlowProblem;
//           using FluidSystem = GetPropType<BaseTypeTag, Properties::FluidSystem>;

//       public:
//           using type = BlackOilTwoPhaseIndices<getPropValue<TypeTag, Properties::EnableSolvent>(),
//                                               getPropValue<TypeTag, Properties::EnableExtbo>(),
//                                               getPropValue<TypeTag, Properties::EnablePolymer>(),
//                                               getPropValue<TypeTag, Properties::EnableEnergy>(),
//                                               getPropValue<TypeTag, Properties::EnableFoam>(),
//                                               getPropValue<TypeTag, Properties::EnableBrine>(),
//                                               /*PVOffset=*/0,
//                                               /*disabledCompIdx=*/FluidSystem::oilCompIdx,
//                                               getPropValue<TypeTag, Properties::EnableMICP>()>;
//       };

//       // SPE11C requires thermal/energy
//       template<class TypeTag>
//       struct EnableEnergy<TypeTag, TTag::FlowSimpleProblem> {
//           static constexpr bool value = true;
//       };

//       // SPE11C requires dispersion
//       template<class TypeTag>
//       struct EnableDispersion<TypeTag, TTag::FlowSimpleProblem> {
//           static constexpr bool value = true;
//       };

//       // Use the simple material law.
//       template<class TypeTag>
//       struct MaterialLaw<TypeTag, TTag::FlowSimpleProblem>
//       {
//       private:
//           using Scalar = GetPropType<TypeTag, Properties::Scalar>;
//           using FluidSystem = GetPropType<TypeTag, Properties::FluidSystem>;

//           using Traits = ThreePhaseMaterialTraits<Scalar,
//                                                   /*wettingPhaseIdx=*/FluidSystem::waterPhaseIdx,
//                                                   /*nonWettingPhaseIdx=*/FluidSystem::oilPhaseIdx,
//                                                   /*gasPhaseIdx=*/FluidSystem::gasPhaseIdx>;
//       public:
//           using EclMaterialLawManager = ::Opm::EclMaterialLawManagerSimple<Traits>;
//           using type = typename EclMaterialLawManager::MaterialLaw;
//       };

//       // Use the TPFA linearizer.
//       template<class TypeTag>
//       struct Linearizer<TypeTag, TTag::FlowSimpleProblem> { using type = TpfaLinearizer<TypeTag>; };

//       template<class TypeTag>
//       struct LocalResidual<TypeTag, TTag::FlowSimpleProblem> { using type = BlackOilLocalResidualTPFA<TypeTag>; };

//       // Diffusion.
//       template<class TypeTag>
//       struct EnableDiffusion<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };

//       template<class TypeTag>
//       struct EnableDisgasInWater<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };

//       template<class TypeTag>
//       struct EnableVapwat<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };

//       // template<class TypeTag>
//       // struct PrimaryVariables<TypeTag, TTag::FlowSimpleProblem> { using type = BlackOilPrimaryVariables<TypeTag, Opm::gpuistl::dense::FieldVector>; };
//   };

// }

// using TypeTag = Opm::Properties::TTag::FlowSimpleProblem;

namespace{
__global__ void instantiate_ad_object(Opm::DenseAd::Evaluation<float, 3>* adObj, double value){
    *adObj = Opm::DenseAd::Evaluation<float, 3>(value, 0);
}

} // END EMPTY NAMESPACE


BOOST_AUTO_TEST_CASE(TestInstantiateADObject)
{
    using Evaluation = Opm::DenseAd::Evaluation<float, 3>;
    double testValue = 123.456;
    Evaluation cpuMadeAd = Evaluation(testValue, 0);

    Evaluation gpuMadeAd; // allocate space for one more AD object on the CPU
    Evaluation *d_ad;

    // allocate space on GPU, run kernel, and move results back to the CPU
    OPM_GPU_SAFE_CALL(hipMalloc(&d_ad, sizeof(Evaluation)));
    instantiate_ad_object<<<1,1>>>(d_ad, testValue);
    OPM_GPU_SAFE_CALL(hipDeviceSynchronize());
    OPM_GPU_SAFE_CALL(hipMemcpy(&gpuMadeAd, d_ad, sizeof(Evaluation), hipMemcpyDeviceToHost));
    OPM_GPU_SAFE_CALL(hipFree(d_ad));

    // Check that the object made in a GPU kernel is equivalent to that made on the CPU
    BOOST_CHECK(cpuMadeAd == gpuMadeAd);
}
