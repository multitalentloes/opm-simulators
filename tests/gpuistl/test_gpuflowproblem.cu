#include "hip/hip_runtime.h"
/*
  Copyright 2025 SINTEF AS
  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <config.h>

#define BOOST_TEST_MODULE TestFlowProblemGpu

#include <boost/test/unit_test.hpp>

#include <opm/material/densead/Evaluation.hpp>
#include <opm/material/fluidmatrixinteractions/EclMaterialLawManagerSimple.hpp>

#include <opm/models/blackoil/blackoilmodel.hh>
#include <opm/models/discretization/common/tpfalinearizer.hh>
#include <opm/models/utils/simulator.hh>

#include <opm/simulators/utils/moduleVersion.hpp>
#include <opm/simulators/flow/FlowProblemBlackoilGpu.hpp>
#include <opm/simulators/flow/FlowProblemBlackoil.hpp>
#include <opm/simulators/flow/FlowProblemBlackoilProperties.hpp>

#include <opm/simulators/flow/BlackoilModelParameters.hpp>
#include <opm/simulators/flow/FlowGenericVanguard.hpp>
#include <opm/simulators/flow/FlowProblemBlackoil.hpp>
#include <opm/simulators/flow/FlowProblemBlackoilProperties.hpp>
#include <opm/simulators/flow/equil/EquilibrationHelpers.hpp>
#include <opm/simulators/linalg/parallelbicgstabbackend.hh>
#include <opm/simulators/linalg/gpuistl/GpuBuffer.hpp>
#include <opm/simulators/linalg/gpuistl/DualBuffer.hpp>
#include <opm/simulators/linalg/gpuistl/GpuView.hpp>
#include <opm/simulators/linalg/gpuistl/gpu_smart_pointer.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpu_safe_call.hpp>
#include <opm/simulators/wells/BlackoilWellModel.hpp>

#include <utility>

#include <hip/hip_runtime.h>

/*
Functionality requested for the blackoil flow problem on gpu:
[X] indicates that the functionality is added and verified with unit test
[-] indicates that the functionality does not seem to be used in spe11
[ ] indicates that the functionality is not added yet

[X] - problem.model().linearizer().getLinearizationType()
[X] - problem.satnumRegionIndex(globalSpaceIdx)
[X] - problem.materialLawParams(globalSpaceIdx)
[X] - problem.rockCompressibility(globalSpaceIdx)
[X] - problem.rockReferencePressure(globalSpaceIdx)
[X] - problem.porosity(globalSpaceIdx, timeIdx)
[-] - problem.maxOilVaporizationFactor(timeIdx, globalSpaceIdx)
[-] - problem.maxGasDissolutionFactor(timeIdx, globalSpaceIdx)
[-] - problem.maxOilSaturation(globalSpaceIdx)
[-] - problem.template rockCompPoroMultiplier<Evaluation>(*this, globalSpaceIdx)
[X] - problem.updateRelperms(mobility_, dirMob_, fluidState_, globalSpaceIdx)
[X] - problem.template rockCompTransMultiplier<Evaluation>(*this, globalSpaceIdx)

*/

#include <opm/material/fluidsystems/BlackOilFluidSystem.hpp>
#include <opm/material/fluidsystems/BlackOilFluidSystemNonStatic.hpp>
#include <opm/material/fluidstates/BlackOilFluidState.hpp>

#include <opm/input/eclipse/Parser/Parser.hpp>
#include <opm/input/eclipse/Deck/Deck.hpp>
#include <opm/input/eclipse/EclipseState/EclipseState.hpp>
#include <opm/input/eclipse/Python/Python.hpp>
#include <opm/input/eclipse/Schedule/Schedule.hpp>

static constexpr const char* deckString1 =
"-- =============== RUNSPEC\n"
"RUNSPEC\n"
"DIMENS\n"
"3 3 3 /\n"
"EQLDIMS\n"
"/\n"
"TABDIMS\n"
"/\n"
"WATER\n"
"GAS\n"
"CO2STORE\n"
"METRIC\n"
"-- =============== GRID\n"
"GRID\n"
"GRIDFILE\n"
"0 0 /\n"
"DX\n"
"27*1 /\n"
"DY\n"
"27*1 /\n"
"DZ\n"
"27*1 /\n"
"TOPS\n"
"9*0 /\n"
"PERMX\n"
"27*1013.25 /\n"
"PORO\n"
"27*0.25 /\n"
"COPY\n"
"PERMX PERMY /\n"
"PERMX PERMZ /\n"
"/\n"
"-- =============== PROPS\n"
"PROPS\n"
"SGWFN\n"
"0.000000E+00 0.000000E+00 1.000000E+00 3.060000E-02\n"
"1.000000E+00 1.000000E+00 0.000000E+00 3.060000E-01 /\n"
"-- =============== SOLUTION\n"
"SOLUTION\n"
"RPTRST\n"
"'BASIC=0' /\n"
"EQUIL\n"
"0 300 100 0 0 0 1 1 0 /\n"
"-- =============== SCHEDULE\n"
"SCHEDULE\n"
"RPTRST\n"
"'BASIC=0' /\n"
"TSTEP\n"
"1 /";

namespace Opm {
  namespace Properties {
      namespace TTag {
          struct FlowSimpleProblem {
              using InheritsFrom = std::tuple<FlowProblem>;
          };
      }

      // Indices for two-phase gas-water.
      template<class TypeTag>
      struct Indices<TypeTag, TTag::FlowSimpleProblem>
      {
      private:
          // it is unfortunately not possible to simply use 'TypeTag' here because this leads
          // to cyclic definitions of some properties. if this happens the compiler error
          // messages unfortunately are *really* confusing and not really helpful.
          using BaseTypeTag = TTag::FlowProblem;
          using FluidSystem = GetPropType<BaseTypeTag, Properties::FluidSystem>;

      public:
          using type = BlackOilTwoPhaseIndices<getPropValue<TypeTag, Properties::EnableSolvent>(),
                                              getPropValue<TypeTag, Properties::EnableExtbo>(),
                                              getPropValue<TypeTag, Properties::EnablePolymer>(),
                                              getPropValue<TypeTag, Properties::EnableEnergy>(),
                                              getPropValue<TypeTag, Properties::EnableFoam>(),
                                              getPropValue<TypeTag, Properties::EnableBrine>(),
                                              /*PVOffset=*/0,
                                              /*disabledCompIdx=*/FluidSystem::oilCompIdx,
                                              getPropValue<TypeTag, Properties::EnableMICP>()>;
      };

      // SPE11C requires thermal/energy
      // template<class TypeTag>
      // struct EnableEnergy<TypeTag, TTag::FlowSimpleProblem> {
      //     static constexpr bool value = true;
      // };

      // SPE11C requires dispersion
      template<class TypeTag>
      struct EnableDispersion<TypeTag, TTag::FlowSimpleProblem> {
          static constexpr bool value = true;
      };

      // Use the simple material law.
      template<class TypeTag>
      struct MaterialLaw<TypeTag, TTag::FlowSimpleProblem>
      {
      private:
          using Scalar = GetPropType<TypeTag, Properties::Scalar>;
          using FluidSystem = GetPropType<TypeTag, Properties::FluidSystem>;
      public:
          using Traits = ThreePhaseMaterialTraits<Scalar,
                                                  /*wettingPhaseIdx=*/FluidSystem::waterPhaseIdx,
                                                  /*nonWettingPhaseIdx=*/FluidSystem::oilPhaseIdx,
                                                  /*gasPhaseIdx=*/FluidSystem::gasPhaseIdx>;
          using EclMaterialLawManager = ::Opm::EclMaterialLawManagerSimple<Traits>;
          using type = typename EclMaterialLawManager::MaterialLaw;
      };

      // Use the TPFA linearizer.
      template<class TypeTag>
      struct Linearizer<TypeTag, TTag::FlowSimpleProblem> { using type = TpfaLinearizer<TypeTag>; };

      template<class TypeTag>
      struct LocalResidual<TypeTag, TTag::FlowSimpleProblem> { using type = BlackOilLocalResidualTPFA<TypeTag>; };

      // Diffusion.
      template<class TypeTag>
      struct EnableDiffusion<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };

      template<class TypeTag>
      struct EnableDisgasInWater<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };

      template<class TypeTag>
      struct EnableVapwat<TypeTag, TTag::FlowSimpleProblem> { static constexpr bool value = true; };
      // template<class TypeTag>
      // struct PrimaryVariables<TypeTag, TTag::FlowSimpleProblem> { using type = BlackOilPrimaryVariables<TypeTag, Opm::gpuistl::dense::FieldVector>; };
  };

}


#include <iostream>
#include <type_traits>
#include <memory>
// #include <dune/common/mpihelper.hh>
#include <dune/common/parallel/mpihelper.hh>
#include <opm/models/utils/start.hh>

template<class ProblemView>
__global__ void satnumFromFlowProblemBlackoilGpu(ProblemView prob, unsigned short* res)
{
  *res = prob.satnumRegionIndex(0);
}

template<class ProblemView>
__global__ void linTypeFromFlowProblemBlackoilGpu(ProblemView prob, Opm::LinearizationType* res)
{
  *res = prob.model().linearizer().getLinearizationType();
}

template<class ProblemView>
__global__ void rockCompressibilityFromFlowProblemBlackoilGpu(ProblemView prob, double* res)
{
  *res = prob.rockCompressibility(0);
}

template<class ProblemView>
__global__ void porosityFromFlowProblemBlackoilGpu(ProblemView prob, double* res)
{
  *res = prob.porosity(0, 0);
}

template<class ProblemView>
__global__ void rockReferencePressureFromFlowProblemBlackoilGpu(ProblemView prob, double* res)
{
  *res = prob.rockReferencePressure(0);
}

template<class ProblemView>
__global__ void materialLawParamsCallable(ProblemView prob)
{
  auto matLawParams = prob.materialLawParams(0);
}

template<class DirMobPtr, class ProblemView, class MobArr, class FluidState>
__global__ void updateRelPermsFromFlowProblemBlackoilGpu(ProblemView prob, MobArr mob, FluidState fs)
{
  auto dirPtr = DirMobPtr(); // produces nullptr, this value is not used in the function, but should match signature
  prob.updateRelperms(mob, dirPtr, fs, 0);
}


BOOST_AUTO_TEST_CASE(TestInstantiateGpuFlowProblem)
{
  using TypeTag = Opm::Properties::TTag::FlowSimpleProblem;
  // FIXTURE FROM TEST EQUIL
  int argc1 = boost::unit_test::framework::master_test_suite().argc;
  char** argv1 = boost::unit_test::framework::master_test_suite().argv;

#if HAVE_DUNE_FEM
  Dune::Fem::MPIManager::initialize(argc1, argv1);
#else
  Dune::MPIHelper::instance(argc1, argv1);
#endif

  using namespace Opm;
  FlowGenericVanguard::setCommunication(std::make_unique<Opm::Parallel::Communication>());
  Opm::ThreadManager::registerParameters();
  BlackoilModelParameters<double>::registerParameters();
  AdaptiveTimeStepping<TypeTag>::registerParameters();
  Parameters::Register<Parameters::EnableTerminalOutput>("Dummy added for the well model to compile.");
  registerAllParameters_<TypeTag>();

  // END OF FIXTURE FROM TEST EQUIL

  using Simulator = Opm::GetPropType<TypeTag, Opm::Properties::Simulator>;

  // TODO: will this actually refer to the very_simple_deck.DATA inside the gpuistl folder,
  // TODO: do we need to keep track of the path since it can be hipified?
  const char* filename = "very_simple_deck.DATA";
  const auto filenameArg = std::string {"--ecl-deck-file-name="} + filename;

  const char* argv2[] = {
      "test_gpuflowproblem",
      filenameArg.c_str(),
      "--check-satfunc-consistency=false",
  };

  Opm::setupParameters_<TypeTag>(/*argc=*/sizeof(argv2)/sizeof(argv2[0]), argv2, /*registerParams=*/false);

  Opm::FlowGenericVanguard::readDeck(filename);

  auto sim = std::make_unique<Simulator>();

  using ThreePhaseTraits = typename GetPropType<TypeTag, Properties::MaterialLaw>::Traits;

  // using ThreePhaseParams = TypeTag::MaterialLaw::EclMaterialLawManager::MaterialLawParams;
  // using ThreePhaseParams = typename GetPropType<TypeTag, Properties::MaterialLaw>::EclMaterialLawManager::MaterialLawParams;
  using ThreePhaseParams = typename ::Opm::EclMaterialLawManagerSimple<ThreePhaseTraits>::MaterialLawParams;
  using CpuGasWaterTwoPhaseLaw = ThreePhaseParams::GasWaterParams;

  enum { waterPhaseIdx = ThreePhaseTraits::wettingPhaseIdx };
  enum { oilPhaseIdx = ThreePhaseTraits::nonWettingPhaseIdx };
  enum { gasPhaseIdx = ThreePhaseTraits::gasPhaseIdx };
  enum { numPhases = ThreePhaseTraits::numPhases };
  using GasWaterTraits = TwoPhaseMaterialTraits<double, waterPhaseIdx, gasPhaseIdx>;

  using GPUBufferInterpolation = Opm::PiecewiseLinearTwoPhaseMaterialParams<GasWaterTraits, Opm::gpuistl::GpuBuffer<double>>;
  using GPUViewInterpolation = Opm::PiecewiseLinearTwoPhaseMaterialParams<GasWaterTraits, Opm::gpuistl::GpuView<double>>;

  auto problemGpuBuf = Opm::gpuistl::copy_to_gpu<double, Opm::gpuistl::GpuBuffer, Opm::gpuistl::DualBuffer, TypeTag, TypeTag>(sim->problem());
  auto problemGpuView = Opm::gpuistl::make_view<Opm::gpuistl::GpuView, Opm::gpuistl::ValueAsPointer>(problemGpuBuf);

  unsigned short satNumOnCpu;
  unsigned short* satNumOnGpu;
  std::ignore = hipMalloc(&satNumOnGpu, sizeof(unsigned short));
  satnumFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, satNumOnGpu);
  std::ignore = hipMemcpy(&satNumOnCpu, satNumOnGpu, sizeof(unsigned short), hipMemcpyDeviceToHost);
  BOOST_CHECK_EQUAL(satNumOnCpu, sim->problem().satnumRegionIndex(0));
  std::ignore = hipFree(satNumOnGpu);

  Opm::LinearizationType linTypeOnCpu;
  Opm::LinearizationType* linTypeOnGpu;
  std::ignore = hipMalloc(&linTypeOnGpu, sizeof(Opm::LinearizationType));
  linTypeFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, linTypeOnGpu);
  std::ignore = hipMemcpy(&linTypeOnCpu, linTypeOnGpu, sizeof(Opm::LinearizationType), hipMemcpyDeviceToHost);
  auto linTypeFromCPUSimulator = sim->problem().model().linearizer().getLinearizationType();
  BOOST_CHECK_EQUAL(linTypeOnCpu.type, linTypeFromCPUSimulator.type);
  std::ignore = hipFree(linTypeOnGpu);

  double rocmCompressibilityOnCpu;
  double* rockCompressibilityOnGpu;
  std::ignore = hipMalloc(&rockCompressibilityOnGpu, sizeof(double));
  rockCompressibilityFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, rockCompressibilityOnGpu);
  std::ignore = hipMemcpy(&rocmCompressibilityOnCpu, rockCompressibilityOnGpu, sizeof(double), hipMemcpyDeviceToHost);
  BOOST_CHECK_EQUAL(rocmCompressibilityOnCpu, sim->problem().rockCompressibility(0));
  std::ignore = hipFree(rockCompressibilityOnGpu);

  double porosityOnCpu;
  double* porosityOnGpu;
  std::ignore = hipMalloc(&porosityOnGpu, sizeof(double));
  porosityFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, porosityOnGpu);
  std::ignore = hipMemcpy(&porosityOnCpu, porosityOnGpu, sizeof(double), hipMemcpyDeviceToHost);
  BOOST_CHECK_EQUAL(porosityOnCpu, sim->problem().porosity(0, 0));
  std::ignore = hipFree(porosityOnGpu);

  double referencePressureOnCpu;
  double* referencePressureOnGpu;
  std::ignore = hipMalloc(&referencePressureOnGpu, sizeof(double));
  rockReferencePressureFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, referencePressureOnGpu);
  std::ignore = hipMemcpy(&referencePressureOnCpu, referencePressureOnGpu, sizeof(double), hipMemcpyDeviceToHost);
  BOOST_CHECK_EQUAL(referencePressureOnCpu, sim->problem().rockReferencePressure(0));
  std::ignore = hipFree(referencePressureOnGpu);

  materialLawParamsCallable<<<1, 1>>>(problemGpuView);

  using FluidSystem = Opm::BlackOilFluidSystem<double>;
  using Evaluation = Opm::DenseAd::Evaluation<double,2>;
  using Scalar = double;
  // using DirectionalMobilityPtr = Utility::CopyablePtr<DirectionalMobility<TypeTag, Evaluation>>;
  using DirectionalMobilityPtr = Utility::CopyablePtr<DirectionalMobility<TypeTag>>;
  
  // Create the fluid system
  Opm::Parser parser;
  auto deck = parser.parseString(deckString1);
  auto python = std::make_shared<Opm::Python>();
  Opm::EclipseState eclState(deck);
  Opm::Schedule schedule(deck, eclState, python);

  FluidSystem::initFromState(eclState, schedule);
  auto& dynamicFluidSystem = FluidSystem::getNonStaticInstance();
  auto dynamicGpuFluidSystemBuffer = ::Opm::gpuistl::copy_to_gpu<::Opm::gpuistl::GpuBuffer, double>(dynamicFluidSystem);
  auto dynamicGpuFluidSystemView = ::Opm::gpuistl::make_view<::Opm::gpuistl::GpuView, ::Opm::gpuistl::ValueAsPointer>(dynamicGpuFluidSystemBuffer);
  auto gpufluidstate = BlackOilFluidState<double, decltype(dynamicGpuFluidSystemView)>(dynamicGpuFluidSystemView);
  // Create MobArr
  double testValue = 0.5;
  // Create an array of Evaluations on CPU
  using MobArr = std::array<Evaluation, 2>;
  MobArr cpuMobArray;
  cpuMobArray[0] = Evaluation(testValue, 0);
  cpuMobArray[1] = Evaluation(testValue, 1);
  
  // Copy to GPU
  MobArr* d_mobArray;
  OPM_GPU_SAFE_CALL(hipMalloc(&d_mobArray, sizeof(MobArr)));
  OPM_GPU_SAFE_CALL(hipMemcpy(d_mobArray, &cpuMobArray, sizeof(MobArr), hipMemcpyHostToDevice));
  
  updateRelPermsFromFlowProblemBlackoilGpu<DirectionalMobilityPtr><<<1, 1>>>(problemGpuView, *d_mobArray, gpufluidstate);
}
