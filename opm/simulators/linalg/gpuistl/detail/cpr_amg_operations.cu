#include "hip/hip_runtime.h"
/*
  Copyright 2025 Equinor ASA

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <config.h>

#include <opm/simulators/linalg/gpuistl/detail/cpr_amg_operations.hpp>
#include <opm/simulators/linalg/gpuistl/detail/deviceBlockOperations.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpuThreadUtils.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpu_safe_call.hpp>

#include <hip/hip_runtime.h>

namespace Opm::gpuistl::detail
{

namespace
{
// Kernel for calculating quasi-IMPES weights
template <typename T>
__global__ void quasiImpesWeightsKernel(const T* matrix,
                                       T* weights,
                                       const int* diagonalIndices,
                                       const int numberOfRows,
                                       const int blockSize,
                                       const int pressureVarIndex,
                                       const bool transpose)
{
    const auto row = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < numberOfRows) {
        const int diagIdx = diagonalIndices[row];
        const int blockOffset = diagIdx * blockSize * blockSize;
        const T* block = matrix + blockOffset;

        // Set up RHS with 1.0 at pressure index
        T rhs[3] = {0}; // Maximum block size is 3
        rhs[pressureVarIndex] = 1.0;

        // Storage for solution
        T bweights[3] = {0}; // Maximum block size is 3

        // Solve the system
        if (transpose) {
            // Solve using original matrix
            solveBlock(block, rhs, bweights, blockSize);
        } else {
            // Create transposed block for solving
            T transposed[9]; // Maximum block size squared is 3x3=9
            transposeBlock(block, transposed, blockSize);
            solveBlock(transposed, rhs, bweights, blockSize);
        }

        // Find maximum absolute value for normalization
        T invMaxAbs = abs(bweights[0]);
        for (int j = 1; j < blockSize; ++j) {
            invMaxAbs = max(invMaxAbs, abs(bweights[j]));
        }
        invMaxAbs = T(1.0) / invMaxAbs;

        // Normalize and store weights
        for (int j = 0; j < blockSize; ++j) {
            weights[row * blockSize + j] = bweights[j] * invMaxAbs;
        }
    }
}

// Kernel to calculate matrix entries for the coarse level - processes each row in parallel
template <typename T>
__global__ void calculateCoarseEntriesKernel(const T* fineNonZeroValues,
                                           T* coarseNonZeroValues,
                                           const T* weights,
                                           const int* rowIndices,
                                           const int* colIndices,
                                           const int numberOfRows,
                                           const int blockSize,
                                           const int pressureVarIndex,
                                           const bool transpose)
{
    // Each thread processes one row of the matrix
    const auto row = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < numberOfRows) {
        // Get start and end indices for this row
        const int start = rowIndices[row];
        const int end = rowIndices[row + 1];

        // Process all non-zeros in this row
        for (int i = start; i < end; i++) {
            const int col = colIndices[i];
            const int blockOffset = i * blockSize * blockSize;
            T matrixEl = 0.0;

            if (transpose) {
                // Use column weight
                const T* bw = weights + col * blockSize;
                for (int j = 0; j < blockSize; ++j) {
                    matrixEl += fineNonZeroValues[blockOffset + pressureVarIndex * blockSize + j] * bw[j];
                }
            } else {
                // Use row weight
                const T* bw = weights + row * blockSize;
                for (int j = 0; j < blockSize; ++j) {
                    matrixEl += fineNonZeroValues[blockOffset + j * blockSize + pressureVarIndex] * bw[j];
                }
            }

            coarseNonZeroValues[i] = matrixEl;
        }
    }
}

// Kernel to restrict a fine vector to a coarse vector
template <typename T>
__global__ void restrictVectorKernel(const T* fine,
                                    T* coarse,
                                    const T* weights,
                                    const int numberOfBlocks,
                                    const int blockSize,
                                    const int pressureVarIndex,
                                    const bool transpose)
{
    const auto blockIndex = blockDim.x * blockIdx.x + threadIdx.x;

    if (blockIndex < numberOfBlocks) {
        T rhsEl = 0.0;

        if (transpose) {
            // Just extract the pressure component
            rhsEl = fine[blockIndex * blockSize + pressureVarIndex];
        } else {
            // Weighted sum of components
            const T* bw = weights + blockIndex * blockSize;
            for (int i = 0; i < blockSize; ++i) {
                rhsEl += fine[blockIndex * blockSize + i] * bw[i];
            }
        }

        coarse[blockIndex] = rhsEl;
    }
}

// Kernel to prolongate a coarse vector to a fine vector
template <typename T>
__global__ void prolongateVectorKernel(const T* coarse,
                                      T* fine,
                                      const T* weights,
                                      const int numberOfBlocks,
                                      const int blockSize,
                                      const int pressureVarIndex,
                                      const bool transpose)
{
    const auto blockIndex = blockDim.x * blockIdx.x + threadIdx.x;

    if (blockIndex < numberOfBlocks) {
        if (transpose) {
            // Distribute the coarse value using weights
            const T* bw = weights + blockIndex * blockSize;
            for (int i = 0; i < blockSize; ++i) {
                fine[blockIndex * blockSize + i] = coarse[blockIndex] * bw[i];
            }
        } else {
            // Only update the pressure component
            fine[blockIndex * blockSize + pressureVarIndex] = coarse[blockIndex];
        }
    }
}
} // anonymous namespace

// Implementation of getQuasiImpesWeights for GPU
template <typename T>
void getQuasiImpesWeights(const GpuSparseMatrix<T>& matrix,
                          std::size_t pressureVarIndex,
                          bool transpose,
                          GpuVector<T>& weights,
                          const GpuVector<int>& diagonalIndices)
{
    const int blockSize = matrix.blockSize();
    const int numberOfRows = matrix.N();

    // Check that block size is supported (1-3)
    if (blockSize < 1 || blockSize > 3) {
        throw std::runtime_error("Unsupported block size for getQuasiImpesWeights: " +
                                std::to_string(blockSize) +
                                ". Only block sizes 1-3 are supported.");
    }

    // Ensure weights vector has the right size
    if (weights.dim() != numberOfRows * blockSize) {
        throw std::runtime_error("Weights vector has incorrect size");
    }

    // Initialize weights to zero
    weights = 0.0;

    // Calculate optimal thread block size for CUDA
    int threadBlockSize = getCudaRecomendedThreadBlockSize(quasiImpesWeightsKernel<T>);
    int nThreadBlocks = getNumberOfBlocks(numberOfRows, threadBlockSize);

    // Launch kernel
    quasiImpesWeightsKernel<<<nThreadBlocks, threadBlockSize>>>(
        matrix.getNonZeroValues().data(),
        weights.data(),
        diagonalIndices.data(),
        numberOfRows,
        blockSize,
        pressureVarIndex,
        transpose
    );
}

template <typename T>
void calculateCoarseEntries(const GpuSparseMatrix<T>& fineMatrix,
                           GpuSparseMatrix<T>& coarseMatrix,
                           const GpuVector<T>& weights,
                           std::size_t pressureVarIndex,
                           bool transpose)
{
    const int blockSize = fineMatrix.blockSize();
    const int numberOfRows = fineMatrix.N();

    int threadBlockSize = getCudaRecomendedThreadBlockSize(calculateCoarseEntriesKernel<T>);
    int nThreadBlocks = getNumberOfBlocks(numberOfRows, threadBlockSize);

    calculateCoarseEntriesKernel<<<nThreadBlocks, threadBlockSize>>>(
        fineMatrix.getNonZeroValues().data(),
        coarseMatrix.getNonZeroValues().data(),
        weights.data(),
        fineMatrix.getRowIndices().data(),
        fineMatrix.getColumnIndices().data(),
        fineMatrix.N(),
        fineMatrix.blockSize(),
        pressureVarIndex,
        transpose
    );
}

template <typename T>
void restrictVector(const GpuVector<T>& fine,
                   GpuVector<T>& coarse,
                   const GpuVector<T>& weights,
                   std::size_t pressureVarIndex,
                   bool transpose)
{
    const int blockSize = fine.dim() / coarse.dim();
    const int numberOfBlocks = coarse.dim();

    int threadBlockSize = getCudaRecomendedThreadBlockSize(restrictVectorKernel<T>);
    int nThreadBlocks = getNumberOfBlocks(numberOfBlocks, threadBlockSize);

    restrictVectorKernel<<<nThreadBlocks, threadBlockSize>>>(
        fine.data(),
        coarse.data(),
        weights.data(),
        numberOfBlocks,
        blockSize,
        pressureVarIndex,
        transpose
    );
}

template <typename T>
void prolongateVector(const GpuVector<T>& coarse,
                     GpuVector<T>& fine,
                     const GpuVector<T>& weights,
                     std::size_t pressureVarIndex,
                     bool transpose)
{
    const int blockSize = fine.dim() / coarse.dim();
    const int numberOfBlocks = coarse.dim();

    int threadBlockSize = getCudaRecomendedThreadBlockSize(prolongateVectorKernel<T>);
    int nThreadBlocks = getNumberOfBlocks(numberOfBlocks, threadBlockSize);

    prolongateVectorKernel<<<nThreadBlocks, threadBlockSize>>>(
        coarse.data(),
        fine.data(),
        weights.data(),
        numberOfBlocks,
        blockSize,
        pressureVarIndex,
        transpose
    );
}

// Explicit template instantiations
template void getQuasiImpesWeights(
    const GpuSparseMatrix<double>& matrix,
    std::size_t pressureVarIndex,
    bool transpose,
    GpuVector<double>& weights,
    const GpuVector<int>& diagonalIndices);

template void getQuasiImpesWeights(
    const GpuSparseMatrix<float>& matrix,
    std::size_t pressureVarIndex,
    bool transpose,
    GpuVector<float>& weights,
    const GpuVector<int>& diagonalIndices);

template void calculateCoarseEntries(const GpuSparseMatrix<double>& fineMatrix,
    GpuSparseMatrix<double>& coarseMatrix,
    const GpuVector<double>& weights,
    std::size_t pressureVarIndex,
    bool transpose);
template void calculateCoarseEntries(
    const GpuSparseMatrix<float>& fineMatrix,
    GpuSparseMatrix<float>& coarseMatrix,
    const GpuVector<float>& weights,
    std::size_t pressureVarIndex,
    bool transpose);

template void restrictVector(
    const GpuVector<double>& fine,
    GpuVector<double>& coarse,
    const GpuVector<double>& weights,
    std::size_t pressureVarIndex,
    bool transpose);
template void restrictVector(
    const GpuVector<float>& fine,
    GpuVector<float>& coarse,
    const GpuVector<float>& weights,
    std::size_t pressureVarIndex,
    bool transpose);

template void prolongateVector(
    const GpuVector<double>& coarse,
    GpuVector<double>& fine,
    const GpuVector<double>& weights,
    std::size_t pressureVarIndex,
    bool transpose);
template void prolongateVector(
    const GpuVector<float>& coarse,
    GpuVector<float>& fine,
    const GpuVector<float>& weights,
    std::size_t pressureVarIndex,
    bool transpose);
} // namespace Opm::gpuistl::detail